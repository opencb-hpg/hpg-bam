#include "hip/hip_runtime.h"

#include <omp.h>

#include "bam.h"
#include "qc_kernel_cuda.h"

extern "C" {
    #include "log.h"
}

//------------------------------------------------------------------------------------
//  kernels call functions
//------------------------------------------------------------------------------------

void call_kernel_basic_stats(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, int* d_strand_counter_p, int* d_map_quality_p,  int* d_alignment_length_p, int num_alignments) {
    kernel_bam_qc_basic_stats <<< dimGrid, dimBlock>>>(d_core_data_p, d_strand_counter_p, d_map_quality_p, d_alignment_length_p, num_alignments);
}

void call_kernel_map_errors(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, uint32_t* d_cigar_data_p, qc_alignment_t* d_qc_alignment_p, int num_alignments) {
    kernel_bam_qc_map_errors <<< dimGrid, dimBlock>>>(d_core_data_p, d_cigar_data_p, d_qc_alignment_p, num_alignments);
}

//------------------------------------------------------------------------------------
//  kernel for bam qc basic statistics ( G P U  implementation)
//------------------------------------------------------------------------------------

__global__ void kernel_bam_qc_basic_stats(bam_data_core_t* d_core_data_p, int* d_strand_counter_p, int* d_map_quality_p, int* d_alignment_length_p, int num_alignments) {

    __shared__ int s_strand_counter_p[OPTIMAL_THREADS_FOR_COMPUTE_CAPABILITY_20];
    __shared__ int s_map_quality_p[OPTIMAL_THREADS_FOR_COMPUTE_CAPABILITY_20];
    __shared__ int s_alignment_length_p[OPTIMAL_THREADS_FOR_COMPUTE_CAPABILITY_20];

    int k;
    unsigned int tid = threadIdx.x;

    k = blockIdx.x * blockDim.x + threadIdx.x;

//   if (threadIdx.x == 0 && blockIdx.x == 0) {
//     int n, s = 0, q = 0;
//     for(n=0 ; n<blockDim.x ; n++) {
//       s += d_bam_data_batch_p[n + k].strand;
//       q += d_bam_data_batch_p[n + k].map_quality;
//     }
//     printf("total: strand (+) = %i, quality = %i\n", s, q);
//   }
//   __syncthreads();

    if (k < num_alignments) {
        s_strand_counter_p[tid] = d_core_data_p[k].strand;
        s_map_quality_p[tid] = d_core_data_p[k].map_quality;
        s_alignment_length_p[tid] = d_core_data_p[k].alignment_length;
        //if (threadIdx.x == 0 && blockIdx.x == 0) { printf("init: bid = %i, tid = %i: counter[0] = %i\n", blockIdx.x, threadIdx.x, s_strand_counter_p[0]);}
    } else {
        s_strand_counter_p[tid] = 0;
        s_map_quality_p[tid] = 0;
        s_alignment_length_p[tid] = 0;
    }
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            s_strand_counter_p[tid] += s_strand_counter_p[tid + s];
            s_map_quality_p[tid] += s_map_quality_p[tid + s];
            s_alignment_length_p[tid] += s_alignment_length_p[tid + s];
        }
        //if (threadIdx.x == 0 && blockIdx.x == 0) { printf("step %i: bid = %i, tid = %i: counter[0] = %i\n", s, blockIdx.x, threadIdx.x, s_strand_counter_p[0]);}
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) {
        d_strand_counter_p[blockIdx.x] = s_strand_counter_p[0];
        d_map_quality_p[blockIdx.x] = s_map_quality_p[0];
        d_alignment_length_p[blockIdx.x] = s_alignment_length_p[0];
        //if (threadIdx.x == 0 && blockIdx.x == 0) { printf("done: bid = %i, tid = %i: counter[0] = %i\n", blockIdx.x, threadIdx.x, s_strand_counter_p[0]);}
    }
}

//------------------------------------------------------------------------------------
//  kernel for bam qc alignment mismatch count ( G P U implementation)
//------------------------------------------------------------------------------------

__global__ void kernel_bam_qc_map_errors(bam_data_core_t* d_core_data_p, uint32_t* d_cigar_data_p, qc_alignment_t* d_qc_alignment_p, int num_alignments) {

    int k;
    int cigar_start_pos, cigar_end_pos, cigar_operation, cigar_num;
    uint32_t cigar_position;

    k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < num_alignments) {
        cigar_start_pos = d_core_data_p[k].cigar_index;
        //cigar_end_pos = cigar_start_pos + 1;
        cigar_end_pos = d_core_data_p[k+1].cigar_index;

        for (int i = cigar_start_pos; i < cigar_end_pos; i++) {
            cigar_position = d_cigar_data_p[i];
            cigar_operation = (cigar_position & BAM_CIGAR_MASK);
            cigar_num = cigar_position >> BAM_CIGAR_SHIFT;
            //printf("cigar operation: %i, cigar num nts: %i\n", cigar_operation, cigar_num);

            switch (cigar_operation) {
                case BAM_CMATCH:
                    d_qc_alignment_p[k].counters[M] += cigar_num;
                    break;  //M
                case BAM_CINS:
                    d_qc_alignment_p[k].counters[I] += cigar_num;
                    break;  //I
                case BAM_CDEL:
                    d_qc_alignment_p[k].counters[D] += cigar_num;
                    break;  //D
                case BAM_CEQUAL:
                    d_qc_alignment_p[k].counters[EQUAL] += cigar_num;
                    break; //=
                case BAM_CDIFF:
                    d_qc_alignment_p[k].counters[X] += cigar_num;
                    break;  //X
            }
        }

        d_qc_alignment_p[k].counters[MISMATCHES] = d_qc_alignment_p[k].counters[D] + d_qc_alignment_p[k].counters[I] + d_qc_alignment_p[k].counters[X];
    }
}
