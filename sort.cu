#include "hip/hip_runtime.h"

#ifndef SORT_CU
#define SORT_CU

#include <stdlib.h>
#include <stdio.h>

#ifdef THRUST-GPU
    #include <thrust/host_vector.h>
    #include <thrust/device_vector.h>
    #include <thrust/sort.h>
    #include <thrust/copy.h>
#endif

extern "C" {
    #include "aligner_dataset.h"
    #include "aligner_dataset_file.h"
    #include "bam.h"
    #include "bam_reader.h"
    #include "bam_writer.h"
    #include "chrom_alignments.h"
    #include "commons.h"
    #include "file_utils.h"
    #include "log.h"
    #include "sam.h"
    #include "sort.h"
    #include "system_utils.h"
}

#include "sort_thrust.h"

#define BLOCK_SIZE 16

/* **********************************************
 *    		Global variables  		*
 * *********************************************/

int bam_reader_alive = 1;

/* ******************************************************
 *    		Function implementations  		*
 * *****************************************************/

void sort_bam_file(size_t batch_size, char* input_filename, char* output_directory) {
    int num_aligments_read;
    char input_shortname[MAX_FULL_PATH_LENGTH];
    char output_filename[MAX_FULL_PATH_LENGTH];
    char** split_filename = (char**) calloc(NUM_OF_CHROMOSOMES, sizeof(char*));

    get_filename_from_path(input_filename, input_shortname);
    sprintf(output_filename, "%s/%s%s", output_directory, input_shortname, SORTED_FILE_SUFFIX);

    alignments_list_t* list_p = alignments_list_new(NUM_OF_CHROMOSOMES);

    // first phase: one-reader vs multi-writers
    // calling threads to read alignments from file and write back to file (one per chromosome)
    bam_reader_t* bam_first_reader_p = bam_reader_new(input_filename, batch_size, 0, list_p, CHROMOSOME_MODE, NO_SORT, ALL_CHROMOSOMES); //base_quality = 0
    bam_writer_t** bam_split_writer_p = (bam_writer_t**) malloc(NUM_OF_CHROMOSOMES * sizeof(bam_writer_t*));

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        split_filename[i] = (char*) malloc(MAX_FULL_PATH_LENGTH * sizeof(char));
        sprintf(split_filename[i], "%s/%s.%i", output_directory, input_shortname, i);
        bam_split_writer_p[i] = bam_writer_new(split_filename[i], list_p, bam_first_reader_p->bam_file_p->bam_header_p, CHROMOSOME_MODE, i);
    }

    bam_reader_start(bam_first_reader_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_writer_start(bam_split_writer_p[i]);
    }

    if (time_flag) {
        start_timer(t1_write);
    }

    num_alignments = bam_reader_join(bam_first_reader_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_writer_join(bam_split_writer_p[i]);
    }

    // second phase: one-reader vs one-writer
    // one-reader is in charged of reading all segmented-files
    alignments_list_free(list_p);
    list_p = alignments_list_new(NUM_OF_CHROMOSOMES);

    bam_reader_t* bam_split_reader_p;

    bam_file_t* bam_file_p = bam_fopen(input_filename);
    bam_writer_t* bam_sorted_writer_p = bam_writer_new(output_filename, list_p, bam_file_p->bam_header_p, SEQUENTIAL_MODE, ALL_CHROMOSOMES);
    bam_fclose(bam_file_p);

    bam_reader_alive = 1;

    bam_writer_start(bam_sorted_writer_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_split_reader_p = bam_reader_new(split_filename[i], batch_size, 0, list_p, SEQUENTIAL_MODE, SORT_BY_POSITION, i);
        bam_reader_start(bam_split_reader_p);
        num_aligments_read = bam_reader_join(bam_split_reader_p);
        //bam_reader_free(bam_split_reader_p);
    }

    bam_reader_alive = 0;

    bam_writer_join(bam_sorted_writer_p);

    // delete bam files per chromosome
    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        remove(split_filename[i]);
        free(split_filename[i]);
    }

    alignments_list_free(list_p);
}

void sort_bam_file_by_id(size_t batch_size, char* input_filename, char* output_directory) {
    int num_aligments_read;
    char input_shortname[MAX_FULL_PATH_LENGTH];
    char output_filename[MAX_FULL_PATH_LENGTH];
    char** split_filename = (char**) calloc(NUM_OF_CHROMOSOMES, sizeof(char*));

    get_filename_from_path(input_filename, input_shortname);
    sprintf(output_filename, "%s/%s%s", output_directory, input_shortname, SORTED_FILE_SUFFIX);

    alignments_list_t* list_p = alignments_list_new(NUM_OF_CHROMOSOMES);

    // first phase: one-reader vs multi-writers
    // calling threads to read alignments from file and write back to file (one per chromosome)
    bam_reader_t* bam_first_reader_p = bam_reader_new(input_filename, batch_size, 0, list_p, CHROMOSOME_MODE, NO_SORT, ALL_CHROMOSOMES); //base_quality = 0
    bam_writer_t** bam_split_writer_p = (bam_writer_t**) malloc(NUM_OF_CHROMOSOMES * sizeof(bam_writer_t*));

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        split_filename[i] = (char*) malloc(MAX_FULL_PATH_LENGTH * sizeof(char));
        sprintf(split_filename[i], "%s/%s.%i", output_directory, input_shortname, i);
        bam_split_writer_p[i] = bam_writer_new(split_filename[i], list_p, bam_first_reader_p->bam_file_p->bam_header_p, CHROMOSOME_MODE, i);
    }

    bam_reader_start(bam_first_reader_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_writer_start(bam_split_writer_p[i]);
    }

    if (time_flag) {
        start_timer(t1_write);
    }

    num_alignments = bam_reader_join(bam_first_reader_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_writer_join(bam_split_writer_p[i]);
    }

    // second phase: one-reader vs one-writer
    // one-reader is in charged of reading all segmented-files
    alignments_list_free(list_p);
    list_p = alignments_list_new(NUM_OF_CHROMOSOMES);

    bam_reader_t* bam_split_reader_p;

    bam_file_t* bam_file_p = bam_fopen(input_filename);
    bam_writer_t* bam_sorted_writer_p = bam_writer_new(output_filename, list_p, bam_file_p->bam_header_p, SEQUENTIAL_MODE, ALL_CHROMOSOMES);
    bam_fclose(bam_file_p);

    bam_reader_alive = 1;

    bam_writer_start(bam_sorted_writer_p);

    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        bam_split_reader_p = bam_reader_new(split_filename[i], batch_size, 0, list_p, SEQUENTIAL_MODE, SORT_BY_POSITION, i);
        bam_reader_start(bam_split_reader_p);
        num_aligments_read = bam_reader_join(bam_split_reader_p);
        //bam_reader_free(bam_split_reader_p);
    }

    bam_reader_alive = 0;
    bam_writer_join(bam_sorted_writer_p);

    // delete bam files per chromosome
    for (int i = 0; i < NUM_OF_CHROMOSOMES; i++) {
        remove(split_filename[i]);
        free(split_filename[i]);
    }

    alignments_list_free(list_p);
}

void sort_dataset_by_id(char* dataset_input, char* output_directory) {
    unsigned int read_lines = 0;
    aligner_dataset_list_t* list_p = aligner_dataset_list_new(20000000);

    aligner_dataset_file_t* dataset_file_p = aligner_dataset_fopen(dataset_input);

    //fill dataset list in the reader
    read_lines = aligner_dataset_read_list(dataset_file_p, list_p, 0);

    //sort dataset
    aligner_dataset_list_sort_by_id(list_p);

    //write sorted file to disk and free resources
    char input_shortname[MAX_FULL_PATH_LENGTH];
    char output_filename[MAX_FULL_PATH_LENGTH];

    get_filename_from_path(dataset_input, input_shortname);
    sprintf(output_filename, "%s/%s%s", output_directory, input_shortname, ALIGNER_DATASET_SORTED_FILE_SUFFIX);

    char** split_filename = (char**) calloc(NUM_OF_CHROMOSOMES, sizeof(char*));
    aligner_dataset_list_write(list_p, output_filename);

    aligner_dataset_list_free(list_p);
    aligner_dataset_fclose(dataset_file_p);
}

#endif /* SORT_CU */
