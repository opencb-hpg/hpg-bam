#include "hip/hip_runtime.h"
/*
 *
 *  Created on: Aug 4, 2011
 *      Author: victor
 */

#ifndef QC_CU_
#define QC_CU_

extern "C" {
    #include "bam_coverage.h"
    #include "bam_data_batch.h"
    #include "bam_data_batch_list.h"
    #include "bam_qc_batch.h"
    #include "bam_qc_report.h"
    #include "bam_reader.h"
    #include "commons.h"    
    #include "file_utils.h"
    #include "gff_data.h"
    #include "gff_reader.h"
    #include "list.h"    
    #include "log.h"
    #include "qc.h"
    #include "qc_hash.h"
    #include "qc_kernel_omp.h"
    #include "sam.h"
    #include "system_utils.h"
}

#include "qc_kernel_cuda.h"

//------------------------------------------------------------------------------------

// global variables for qc process

// bam_qc_batch_list_t bam_qc_batch_list;
list_t bam_qc_batch_list;

int bam_batch_reader_alive = 1;
int gpus_thread_alive = 1;
int cpus_thread_alive = 1;

pthread_mutex_t gpus_thread_alive_lock = PTHREAD_MUTEX_INITIALIZER;
pthread_mutex_t cpus_thread_alive_lock = PTHREAD_MUTEX_INITIALIZER;

//====================================================================================
// thread functions
//====================================================================================

// structures for gpu server thread
//
typedef struct qc_calc_server_input {
  int num_gpu_devices;
  int cpu_num_threads;
  int gpu_device_id[256];
  int nb_reads_per_batch;
  int gpu_num_blocks;
  int gpu_num_threads;
  list_t* gpu_batch_list_p;
  list_t* cpu_batch_list_p;
} qc_calc_server_input_t;

// structures for cpu server thread
//
typedef struct cpus_server_input {
  int cpu_num_threads;
  int max_distance_size;
  list* cpu_batch_list_p;
  qc_mapping_counter_t* qc_mapping_counter;
  char* gff_filename;
  char* output_directory;
  char* input_filename;
} cpus_server_input_t;

// structures for results server thread
//
typedef struct results_server_input {
  int gpu_num_blocks;
  int gpu_num_threads;
  int base_quality;
  qc_mapping_counter_t* qc_mapping_counter;
  char* filename;
  char* report_directory;
} results_server_input_t;

// threads functions

void* qc_calc_server(void* params_p);
void* cpus_server(void* params_p);
void* results_server(void* params_p);


//-----------------------------------------------------
// qc_calc_server,
//
// this thread gets bam data from the bam data batch list,
// copy them to GPU if exists, execute kernel and insert the 
// results into the qc batch list. If not exist, the
// same calculations are performed in CPU using OMP
//-----------------------------------------------------

extern void call_kernel_basic_stats(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, qc_info_t* d_qc_info_p, int* d_strand_counter_p, int* d_alignment_length_p, int* d_map_quality_p, int num_alignments);
extern void call_kernel_map_errors(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, uint32_t* d_cigar_data_p, qc_alignment_t* d_qc_alignment_p, int num_alignments);

void* qc_calc_server(void* params_p) {
	
  LOG_DEBUG("Thread-GPU: START\n");
  
  if (time_flag) { start_timer(t1_qc_calc_server); }
  
  qc_calc_server_input_t* input_p = (qc_calc_server_input_t*) params_p;
  
  int cpu_num_threads = input_p->cpu_num_threads;
  list_item_t* bam_data_batch_list_item_p = NULL;
  list_t* gpu_batch_list_p = input_p->gpu_batch_list_p;
  list_t* cpu_batch_list_p = input_p->cpu_batch_list_p;
  bam_data_batch_t* bam_data_batch_p = NULL;
  bam_qc_batch_t* bam_qc_batch_p = NULL;
  
  // variables for store output results in both CPU and GPU
  //
  qc_info_t* qc_info_p;
  qc_alignment_t* qc_alignment_p;
  int* strand_counter_p;
  int* map_quality_p;
  int* alignment_length_p;

  bam_data_core_t* d_core_data_p;
  uint32_t* d_cigar_data_p;
  qc_info_t* d_qc_info_p;
  qc_alignment_t* d_qc_alignment_p;
  int* d_strand_counter_p;
  int* d_map_quality_p;
  int* d_alignment_length_p;
  
  // selecting GPU device
  //  
  CUDA_SAFE_CALL( hipSetDevice(input_p->gpu_device_id[0]) );	
  
  int reads_alive;
  
  //reads_alive = bam_data_batch_list_get_producers(gpu_batch_list_p);
  reads_alive = list_get_writers(gpu_batch_list_p);
  
  
  //bam_data_batch_list_print(batch_list_p);
  
  //bam_data_batch_list_item_p = bam_data_batch_list_remove(gpu_batch_list_p);
  bam_data_batch_list_item_p = list_remove_item(gpu_batch_list_p);

  while (reads_alive>0 || bam_data_batch_list_item_p!=NULL) {
    LOG_DEBUG("Thread-GPU: waiting for batch....\n");
    //printf("reads_alive: %i, bam_data_batch_list_item_p is NULL: %i\n", reads_alive, (bam_data_batch_list_item_p == NULL));
    
    if (bam_data_batch_list_item_p==NULL) {
      //printf("Thread-GPU: waiting 0.1 s for reads....\n");
      
      sched_yield();
      
      // Delay for a bit
      //struct timespec ts;
      //ts.tv_sec = 0;
      //ts.tv_nsec = 100000000;
      //nanosleep (&ts, NULL);
      
      usleep(10000);
      
    } else {
printf("1 ---->\n");      
     
      //if (time_flag) { start_timer(t1_gpu); }
      
      if ((time_flag) && (gpus_standby_time == 0.0)) { stop_timer(t1_active_reader, t1_active_gpus, gpus_standby_time); }

      char log_message[50];
      sprintf(log_message, "Thread-GPU: processing for batch %i....\n", bam_data_batch_list_item_p->id);
      LOG_DEBUG(log_message);

      number_of_batchs++;
 
      // allocation memory for output results
      //
      qc_info_p = (qc_info_t*) calloc(1, sizeof(qc_info_t));
      
      //int num_alignments = bam_data_batch_list_item_p->data_p->num_alignments;
      bam_data_batch_p = (bam_data_batch_t*) bam_data_batch_list_item_p->data_p;
      int num_alignments = bam_data_batch_p->num_alignments;      
      int num_blocks;
printf("2 ---->\n");

      if (cpu_num_threads == 0) {		// GPU implementation

	num_blocks = (num_alignments / input_p->gpu_num_threads) + 1;
	
	dim3 dimBlock(input_p->gpu_num_threads, 1, 1);
	dim3 dimGrid(num_blocks, 1, 1);

	strand_counter_p = (int*) calloc(num_blocks, sizeof(int));
	map_quality_p = (int*) calloc(num_blocks, sizeof(int));
	alignment_length_p = (int*) calloc(num_blocks, sizeof(int));
	qc_alignment_p = (qc_alignment_t*) calloc(bam_data_batch_p->num_alignments, sizeof(qc_alignment_t));
printf("3 ---->\n");
printf("3.0 ---->\n");
	//printf("bam data batch id %i, num_alignments: %i\n", bam_data_batch_list_item_p->id, bam_data_batch_list_item_p->batch_p->num_alignments);
	
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_core_data_p, (unsigned int) (num_alignments + 1) * sizeof(bam_data_core_t), 0) );
printf("3.1 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_qc_info_p, (unsigned int) sizeof(qc_info_t), 0) );
printf("3.2 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_strand_counter_p, (unsigned int) num_blocks * sizeof(int), 0) );
printf("3.3 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_map_quality_p, (unsigned int) num_blocks * sizeof(int), 0) );
printf("3.4 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_alignment_length_p, (unsigned int) num_blocks * sizeof(int), 0) );
printf("3.5 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_cigar_data_p, (unsigned int) bam_data_batch_p->num_cigar_operations * sizeof(uint32_t), 0) );
printf("3.6 ---->\n");
	CUDA_SAFE_CALL( hipHostAlloc((void**) &d_qc_alignment_p, (unsigned int) num_alignments * sizeof(qc_alignment_t), 0) );
printf("4 ---->\n");
	//printf("memory usage: data_size = %.2f MB, data_indices_size = %.2f MB, gpu_result = %.2f MB, gpu_kmers = %.2f MB\n", fastq_batch_list_item_p->batch_p->data_size / 1e6, fastq_batch_list_item_p->batch_p->data_indices_size / 1e6,  ((unsigned int) fastq_batch_list_item_p->batch_p->num_reads * sizeof(qc_read_t)) / 1e6, ((unsigned int) fastq_batch_list_item_p->batch_p->num_reads * KMERS_COMBINATIONS * sizeof(qc_kmers_t)) / 1e6);
	
	CUDA_SAFE_CALL( hipMemset((void*) d_qc_info_p, 0, (unsigned int) sizeof(qc_info_t)) );

	CUDA_SAFE_CALL( hipMemcpy(d_core_data_p, bam_data_batch_p->core_data_p, (num_alignments + 1) * sizeof(bam_data_core_t), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_cigar_data_p, bam_data_batch_p->cigar_data_p, bam_data_batch_p->num_cigar_operations * sizeof(uint32_t), hipMemcpyHostToDevice) );
	
	CUDA_START_TIMER();
	call_kernel_basic_stats(dimGrid, dimBlock, d_core_data_p, d_qc_info_p, d_strand_counter_p, d_map_quality_p, d_alignment_length_p, num_alignments);
	call_kernel_map_errors(dimGrid, dimBlock, d_core_data_p, d_cigar_data_p, d_qc_alignment_p, num_alignments);
	CUDA_STOP_TIMER();
printf("5 ---->\n");	
  //       for (int j=0; j < bam_data_batch_list_item_p->batch_p->num_cigar_operations; j++) {
  // 	printf("cigar operation: %i, num nts: %i\n", (bam_data_batch_list_item_p->batch_p->cigar_data_p[j])&BAM_CIGAR_MASK, (bam_data_batch_list_item_p->batch_p->cigar_data_p[j])>>BAM_CIGAR_SHIFT);
  //       }
 
	// copy result from GPU (GPU -> CPU)
	//
	CUDA_SAFE_CALL( hipMemcpy(qc_info_p, d_qc_info_p,  sizeof(qc_info_t), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(strand_counter_p, d_strand_counter_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(map_quality_p, d_map_quality_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(alignment_length_p, d_alignment_length_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(qc_alignment_p, d_qc_alignment_p, num_alignments * sizeof(qc_alignment_t), hipMemcpyDeviceToHost) );
printf("6 ---->\n");
	// free memory
	//
	CUDA_SAFE_CALL( hipHostFree(d_core_data_p) );
	CUDA_SAFE_CALL( hipHostFree(d_cigar_data_p) );
	CUDA_SAFE_CALL( hipHostFree(d_qc_info_p) );
	CUDA_SAFE_CALL( hipHostFree(d_strand_counter_p) );
	CUDA_SAFE_CALL( hipHostFree(d_map_quality_p) );
	CUDA_SAFE_CALL( hipHostFree(d_qc_alignment_p) );

      } else {
	
	// accumulation of partial results is only made once
	num_blocks = 1;
	
	strand_counter_p = (int*) calloc(num_blocks, sizeof(int));
	map_quality_p = (int*) calloc(num_blocks, sizeof(int));
	alignment_length_p = (int*) calloc(num_blocks, sizeof(int));
	qc_alignment_p = (qc_alignment_t*) calloc(bam_data_batch_p->num_alignments, sizeof(qc_alignment_t));

	if (time_flag) { start_timer(t1_gpu); }
	cpu_bam_qc_basic_stats(bam_data_batch_p->core_data_p, strand_counter_p, map_quality_p, alignment_length_p, num_alignments, cpu_num_threads);
	cpu_bam_qc_map_errors(bam_data_batch_p->core_data_p, bam_data_batch_p->cigar_data_p, qc_alignment_p, num_alignments);
	if (time_flag) { stop_timer(t1_gpu, t2_gpu, gpu_time); }
      }
printf("7 ---->\n");
	//if (time_flag) { stop_timer(t1_gpu, t2_gpu, gpu_time); }
       
      // create a new qc_batch object
      //
      bam_qc_batch_p = (bam_qc_batch_t*) malloc(sizeof(bam_qc_batch_t));
      bam_qc_batch_p->id = bam_data_batch_list_item_p->id;
      bam_qc_batch_p->num_alignments = bam_data_batch_p->num_alignments;
      bam_qc_batch_p->num_blocks = num_blocks;
      bam_qc_batch_p->qc_info_p = qc_info_p;
      bam_qc_batch_p->qc_alignment_p = qc_alignment_p;
      bam_qc_batch_p->strand_counter_p = strand_counter_p;
      bam_qc_batch_p->map_quality_p = map_quality_p;
      bam_qc_batch_p->alignment_length_p = alignment_length_p;
      //bam_qc_batch_p->alignments_p = bam_data_batch_list_item_p->alignments_p;
printf("8 ---->\n");
      // and insert it into the bam_qc_batch_list
      //
      //bam_qc_batch_list_insert(bam_qc_batch_p, &bam_qc_batch_list);      
      list_item_t* item_p = list_item_new(bam_qc_batch_p->id, 0, bam_qc_batch_p);
      list_insert_item(item_p, &bam_qc_batch_list);
printf("-----> inserting in bam_qc_batch_list id: %i ---->\n", bam_qc_batch_p->id);
      
      //qc_batch_list_print(&qc_batch_list);
    
      // copy the the current batch item to the cpu batch list in order to perform CPU qc operations
      //bam_data_batch_list_insert(bam_data_batch_list_item_p, cpu_batch_list_p);
      list_insert_item(bam_data_batch_list_item_p, cpu_batch_list_p);
      //bam_data_batch_list_item_free(bam_data_batch_list_item_p, true);
printf("-----> inserting in cpu_batch_list_p ---->\n");
      sprintf(log_message, "Thread-GPU:...processing for batch %i done !\n", bam_data_batch_list_item_p->id);
      LOG_DEBUG(log_message);

      //if (time_flag) { stop_timer(t1_gpu, t2_gpu, gpu_time); }
    } // end if-else
    
    // ask again for reads server status
    //		
    //if (time_flag) { start_timer(t1_gpu); }
    //reads_alive = bam_data_batch_list_get_producers(gpu_batch_list_p);
    reads_alive = list_get_writers(gpu_batch_list_p);

    // next batch...: the first in the list
    //    
    //bam_data_batch_list_item_p = bam_data_batch_list_remove(gpu_batch_list_p);
    bam_data_batch_list_item_p = list_remove_item(gpu_batch_list_p);
      
    //if (time_flag) { stop_timer(t1_gpu, t2_gpu, gpu_time); }
  } // end of external while loop

  pthread_mutex_lock(&gpus_thread_alive_lock);
  gpus_thread_alive--;
  pthread_mutex_unlock(&gpus_thread_alive_lock);
  
  //bam_data_batch_list_decr_producers(cpu_batch_list_p);
  list_decr_writers(cpu_batch_list_p);
  list_decr_writers(&bam_qc_batch_list);

  if (time_flag) { stop_timer(t1_qc_calc_server, t2_qc_calc_server, qc_calc_server_time); }
  LOG_DEBUG("Thread-GPU: END\n");
  
  // exiting....
  //
  pthread_exit(0);
}

//---------------------------------------------------------
// cpus_server,
//
// this thread gets bam data from the bam data batch list,
// and calculates number of duplicated alignments,  
// paired end distance and coverage
//---------------------------------------------------------

void* cpus_server(void* params_p) {

  double coverage_time = 0.0;
  struct timeval t1_coverage, t2_coverage;
  
  LOG_DEBUG("Thread-CPU: START\n");
  
  if (time_flag) { start_timer(t1_cpus_server); }
  
  //initialize str_coverage_matrix
  str_coverage_matrix_init();
  
  cpus_server_input_t* input_p = (cpus_server_input_t*) params_p;
  
  qc_mapping_counter_t* qc_mapping_counter_p = (qc_mapping_counter_t*) input_p->qc_mapping_counter;
  int max_distance_size = input_p->max_distance_size;
  int cpu_num_threads =  input_p->cpu_num_threads;
  //bam_data_batch_list_item_t* bam_data_batch_list_item_p = NULL;
  //bam_data_batch_list_t* cpu_batch_list_p = input_p->cpu_batch_list_p;
  bam_data_batch_t* bam_data_batch_p = NULL;
  list_item_t* bam_data_batch_list_item_p = NULL;
  list_t* cpu_batch_list_p = input_p->cpu_batch_list_p;  
  char* gff_filename = input_p->gff_filename;
  char* output_directory = input_p->output_directory;
  char* input_filename = input_p->input_filename;

  // delete previous coverage file to append new data
  bam_coverage_counter_delete_file(output_directory, input_filename);

  // variables for store intermediate and output results in both CPU and GPU
  //
  qc_hash_t* qc_hash_p = (qc_hash_t*) qc_hash_new(QC_HASH_LENGTH);
  
  // variables for coverage (regions data)
  //
  bam_chromosome_coverage_t bam_chromosome_coverage[NUM_OF_CHROMOSOMES];

  int j;
  for (j=0; j < NUM_OF_CHROMOSOMES; j++) {
    bam_chromosome_coverage_init(&bam_chromosome_coverage[j]); 
  }

  gff_data_t* gff_data_p = gff_data_new(gff_filename);
  //printf("num_gff_lines: %i\n", (gff_data_p == NULL) ? 0 : gff_data_p->num_regions);

  //gff_data_print_lines(gff_data_p);
  //gff_data_print_regions(gff_data_p);

  int gpus_alive;

  //gpus_alive = bam_data_batch_list_get_producers(cpu_batch_list_p);
  gpus_alive = list_get_writers(cpu_batch_list_p);
  
  //bam_data_batch_list_print(cpu_batch_list_p);
  
  //bam_data_batch_list_item_p = bam_data_batch_list_remove(cpu_batch_list_p);
  bam_data_batch_list_item_p = list_remove_item(cpu_batch_list_p);
  //bam_data_batch_list_item_p = NULL;
  
  // ---------------- T E S T ------------------
  int test_counter_alignments = 0;

  while (gpus_alive>0 || bam_data_batch_list_item_p!=NULL) {
    //printf("Thread-CPU: waiting for batch....\n");
    //printf("gpus_alive: %i, bam_data_batch_list_item_p is NULL: %i\n", gpus_alive, (bam_data_batch_list_item_p == NULL));

//     if (bam_data_batch_list_item_p==NULL) {
//       //printf("Thread-CPU: waiting 1 s for reads....\n");
// 
//       sched_yield();
//       
//       // Delay for a bit
//       //struct timespec ts;
//       //ts.tv_sec = 0;
//       //ts.tv_nsec = 100000000;
//       //nanosleep (&ts, NULL);
//       
//       usleep(10000);     	    
//     } else {
//        
//       if ((time_flag) && (cpus_standby_time == 0.0)) { stop_timer(t1_active_reader, t1_active_cpus, cpus_standby_time); }
// 		
// 	char log_message[50];
// 	sprintf(log_message, "Thread-CPU: processing for batch %i....\n", bam_data_batch_list_item_p->id);
// 	LOG_DEBUG(log_message);	
// 
// 	// allocation memory for output results
// 	//
// 	bam_data_batch_p = (bam_data_batch_t*) bam_data_batch_list_item_p->data_p;
// 	int num_alignments = bam_data_batch_p->num_alignments;
// 	int cpu_num_threads = input_p->cpu_num_threads;
// 
// 	if (time_flag) { start_timer(t1_cpu); }
// 	
// 	char* id_seq;
// 	int tid, start_coordinate, seq_length;
// 	short int paired_end;
// 	bam_data_core_t* core_data_p;
// 
// 	for (int i=0; i < bam_data_batch_p->num_alignments; i++) {
// 	  id_seq = &(bam_data_batch_p->id_seq_data_p[bam_data_batch_p->core_data_p[i].id_seq_index]);
// 	  
// 	  // ---------------- T E S T ------------------
// 	  //test_counter_alignments++;
// 	  //char* id_seq_test = (char*) calloc(strlen(id_seq) + 2, sizeof(char));
// 	  //strcpy(id_seq_test, id_seq);
// 	  //strcat(id_seq_test, (i%2) == 1 ? "/1" : "/2");
// 	  //printf("id_seq_test: %s\n", id_seq_test);	
// 	  // ---------------- T E S T ------------------
// 	  
// 	  core_data_p = &(bam_data_batch_p->core_data_p[i]);
// 
// 	  tid = core_data_p->chromosome;
// 	  start_coordinate = core_data_p->start_coordinate;
// 	  seq_length = core_data_p->alignment_length;
// 	  paired_end = core_data_p->paired_end;
// 
// 	  //printf("id seq: %s\n", id_seq);
// 	  
// 	  qc_hash_insert_alignment(qc_hash_p, id_seq, tid, start_coordinate, seq_length, paired_end);
// 
// 	  // ---------------- T E S T ------------------	
// 	  //qc_hash_insert_alignment(qc_hash_p, id_seq_test, tid, start_coordinate, seq_length);
// 	  // ---------------- T E S T ------------------	  
// 	}
// 
// 	//if (time_flag) { start_timer(t1_coverage); }
// 	
// 	if (gff_data_batch_in_region(bam_data_batch_p, gff_data_p) != 0) {
// 	    //printf("\nstart_position: %i\n", bam_data_batch_list_item_p->batch_p->start_positions[0]);
// 	    //printf("end_position: %i\n", bam_data_batch_list_item_p->batch_p->last_alignments_position);
// 	    //printf("chromosome: %i, region_start: %i, region_end: %i\n", gff_data_p->gff_regions_p[gff_data_p->actual_region].chromosome, gff_data_p->gff_regions_p[gff_data_p->actual_region].start, gff_data_p->gff_regions_p[gff_data_p->actual_region].end);
// 	    //printf("gff_data_batch_in_region: %i\n", gff_data_batch_in_region(bam_data_batch_list_item_p->batch_p, gff_data_p));
// 	  bam_coverage_compute(bam_data_batch_p, bam_chromosome_coverage, gff_data_p, output_directory, input_filename, cpu_num_threads);
// 	}
// 	
// 	//if (time_flag) { stop_timer(t1_coverage, t2_coverage, coverage_time); }
// 	//printf("total coverage time       (s): \t%10.5f\n", 0.000001 * coverage_time);
// 	//printf("write time                (s): \t%10.5f\n", 0.000001 * write_time);
//       
// 	if (time_flag) { stop_timer(t1_cpu, t2_cpu, cpu_time); }
// 
// 	sprintf(log_message, "Thread-CPU:...processing for batch %i done !\n", bam_data_batch_list_item_p->id);
// 	LOG_DEBUG(log_message);
// 
// 	// F R E E  R E S O U R C E S
// printf("5 ---->\n"); 
// 	// free the current batch item if all processing with the batch is performed
// 	//bam_data_batch_list_item_free(bam_data_batch_list_item_p, true);
// 	bam_data_batch_free((bam_data_batch_t*) bam_data_batch_list_item_p->data_p);
// 	list_item_free(bam_data_batch_list_item_p);
// 	//printf("Thread-CPU:...processing for batch %i done !\n", qc_batch_p->id); 
// 	
// 
//       
//       } // end if-else
//  
      // ask again for reads server status
      //		
      //gpus_alive = bam_data_batch_list_get_producers(cpu_batch_list_p);
      gpus_alive = list_get_writers(cpu_batch_list_p);
printf("6 ---->\n");    
      // next batch...: the first in the list
      //
      //bam_data_batch_list_item_p = bam_data_batch_list_remove(cpu_batch_list_p);
      bam_data_batch_list_item_p = list_remove_item(cpu_batch_list_p);
printf("7 ---->\n");
  } // end of external while loop

  // ---------------- T E S T ------------------	
    
    /*qc_hash_list_t* list_p;
    printf("qc_hash_p->length: %i\n", qc_hash_p->length);
    for (int l=0; l < qc_hash_p->length; l++) {
      list_p = &(qc_hash_p->qc_hash_list_p[l]);
      if (list_p->length == 0) continue;
      printf("list-length: %i\n", list_p->length);
    }*/
    
  // ---------------- T E S T ------------------	
  
  // print the last counters 
  bam_coverage_counter_mark_to_print(bam_chromosome_coverage, true);
  bam_coverage_counter_print(bam_chromosome_coverage, output_directory, input_filename);

  //qc_hash_list_print(qc_hash_p->qc_hash_list_p);

// ---------------- T E S T ------------------	
//   int list_length = 0;
//   int num_lists = 0;
//   int min_list_length = 1;
//   int max_list_length = 1;
//   int mean_list_length = 0;  
//   int count_alignments = 0;
//   
//   qc_hash_list_item_t* item_aux_p;
//   
//   for (int j=0; j < qc_hash_p->length; j++) {
//     //count_alignments += qc_hash_p->qc_hash_list_p[j].length;
//     
//     item_aux_p = qc_hash_p->qc_hash_list_p[j].first_p;
//     list_length = qc_hash_p->qc_hash_list_p[j].length;
//     
//     mean_list_length += list_length;
//     
//     if (list_length > 0) { num_lists++; }
//     
//     if ((list_length < min_list_length) && (list_length != 0)) {
//       min_list_length = list_length;      
//     }
// 
//     if (list_length > max_list_length)  {
//       max_list_length = list_length;
//     }
// 
//     for (int k=0; k < qc_hash_p->qc_hash_list_p[j].length; k++) {
//       count_alignments += item_aux_p->num_pairends1;
//       count_alignments += item_aux_p->num_pairends2;
//       item_aux_p = item_aux_p->next_p;
//     }
//   }
//   printf("\ncount_alignments: %i\n", count_alignments);
//   //printf("test_counter_alignments: %i\n\n", test_counter_alignments);
//   printf("num of lists: %i\n", num_lists);
//   printf("min list length: %i\n", min_list_length);
//   printf("mean list length: %f\n", 1.0 * mean_list_length / num_lists);
//   printf("max list length: %i\n", max_list_length);
// ---------------- T E S T ------------------	  
  
  //calculate over the qc hash table to obtain:
  //    - Mean distance between paired ends
  //    - Histogram of mappings per reads
  
  //unsigned int num_mappings_histogram[MAX_MAPPING_COUNT_IN_HISTOGRAM + 2];
  unsigned long mean_paired_end_distance = 0;
      
  if (time_flag) { start_timer(t1_cpu); }
  //qc_hash_perform_calculations(qc_hash_p, num_mappings_histogram, &mean_paired_end_distance, max_distance_size, cpu_num_threads);
  qc_hash_perform_calculations(qc_hash_p, qc_mapping_counter_p, &mean_paired_end_distance, max_distance_size, cpu_num_threads); //--

  //memcpy(qc_mapping_counter_p->num_mappings_histogram, num_mappings_histogram, sizeof(num_mappings_histogram));  
  
  //int sum_map = 0;
  //for (int i=1; i<=(MAX_MAPPING_COUNT_IN_HISTOGRAM + 1); i++) {
  //  sum_map += (qc_mapping_counter_p->num_mappings_histogram[i] * i);
  //}

  //printf("mean_paired_end_distance: %li, sum_map: %i\n", mean_paired_end_distance, sum_map);

  //qc_mapping_counter_p->mean_paired_end_distance = mean_paired_end_distance / sum_map;
  qc_mapping_counter_p->mean_paired_end_distance = mean_paired_end_distance;
  if (time_flag) { stop_timer(t1_cpu, t2_cpu, cpu_time); }
  
  //free qc hash structure, gff data and chromosome coverage
  for (j=0; j < NUM_OF_CHROMOSOMES; j++) {
    bam_chromosome_coverage_clear(&bam_chromosome_coverage[j]); 
  }
  
  //bam_chromosome_coverage_clear(&bam_chromosome_coverage);
//   for (j=0; j < NUM_OF_CHROMOSOMES; j++) {
//     free(&bam_chromosome_coverage[j]); 
//   }

  qc_hash_free(qc_hash_p, true);
  gff_data_free(gff_data_p);  
    
  // --------------- D E B U G ----------------
  
  printf("--------------- D E B U G ----------------\n");
  
  for (int i=0; i<=(MAX_MAPPING_COUNT_IN_HISTOGRAM + 1); i++) {
    printf("qc_mapping_counter_p->num_mappings_histogram[%i]: %i\n", i, qc_mapping_counter_p->num_mappings_histogram[i]);
  }
  printf("mean_paired_end_distance: %ld\n\n", qc_mapping_counter_p->mean_paired_end_distance);
  //printf("mean_paired_end_distance: %ld\n\n", mean_paired_end_distance / sum_map);
  printf("--------------- D E B U G ----------------\n");
  
  // --------------- D E B U G ----------------
 
  pthread_mutex_lock(&cpus_thread_alive_lock);
  cpus_thread_alive--;
  pthread_mutex_unlock(&cpus_thread_alive_lock);

  if (time_flag) { stop_timer(t1_cpus_server, t2_cpus_server, cpus_server_time); }
  
  LOG_DEBUG("Thread-CPU: END\n");  

  // exiting....
  //
  pthread_exit(0);
}

//---------------------------------------------------------
// results_server,
//
// this thread gets qc results from the qc batch list,
// and makes the post calculations needed for intermediate  
// results. It also launches the reports generation
//---------------------------------------------------------

void* results_server(void* params_p) {

  LOG_DEBUG("Thread-RESULTS: START\n");
  
  if (time_flag) { start_timer(t1_results_server); }

  results_server_input_t* input_p = (results_server_input_t*) params_p;
 
  // variables for storing qc report information
  //
  bam_qc_report_t bam_qc_report;
  memset(&bam_qc_report, 0, sizeof(bam_qc_report_t));

  qc_mapping_counter_t* qc_mapping_counter_p = (qc_mapping_counter_t*) input_p->qc_mapping_counter;  
  int nb_total_threads = input_p->gpu_num_blocks * input_p->gpu_num_threads;
  int base_quality = input_p->base_quality;
  int i, alignments;
    
  // go through the results_batch list, and process it
  // take and remove the first item, and so on...
  //
  list_item_t* item_p = NULL;
  bam_qc_batch_t* bam_qc_batch_p = NULL;
  int gpus_alive, cpus_alive;
  
  // getting gpus thread status
  //
  pthread_mutex_lock(&gpus_thread_alive_lock);
  gpus_alive = gpus_thread_alive;
  pthread_mutex_unlock(&gpus_thread_alive_lock);

  pthread_mutex_lock(&cpus_thread_alive_lock);
  cpus_alive = cpus_thread_alive;
  pthread_mutex_unlock(&cpus_thread_alive_lock);
  
  //bam_qc_batch_list_print(&bam_qc_batch_list);
  
  // get the first element in the list
  //
  //bam_qc_batch_p = bam_qc_batch_list_remove(&bam_qc_batch_list);
  item_p = list_remove_item(&bam_qc_batch_list);
  bam_qc_batch_p = (bam_qc_batch_t*) item_p->data_p;  
  
  //tic("----> processing qc_batch");
  while (gpus_alive>0 || cpus_alive>0) {
    //printf("while... gpus_alive: %i, cpus_alive: %i, bam_qc_batch_p is NULL: %i\n", gpus_alive, cpus_alive, (bam_qc_batch_p == NULL) ? 1:0);
    if (bam_qc_batch_p==NULL) {
      //printf("Thread-RESULTS: waiting 1 s for GPU outputs....\n");
      
      sched_yield();
      
      // Delay for a bit
      //struct timespec ts;
      //ts.tv_sec = 0;
      //ts.tv_nsec = 100000000;
      //nanosleep (&ts, NULL);
      
      usleep(10000);
	    
    } else {
      
      //number_of_batchs++;
      if ((time_flag) && (results_standby_time == 0.0)) { stop_timer(t1_active_reader, t1_active_results, results_standby_time); }
      if (time_flag) { start_timer(t1_result); }
	      
      char log_message[50];
      sprintf(log_message, "Thread-RESULTS: processing for bam batch %i....\n", bam_qc_batch_p->id);
      LOG_DEBUG(log_message);

      // result processing batch per batch
      alignments = bam_qc_batch_p->num_alignments;
      bam_qc_report.num_alignments += alignments;
     
      for (int k=0; k<bam_qc_batch_p->num_blocks; k++) {
	bam_qc_report.strand_counter += bam_qc_batch_p->strand_counter_p[k];
	bam_qc_report.mean_map_quality += bam_qc_batch_p->map_quality_p[k];
	bam_qc_report.mean_alignment_length += bam_qc_batch_p->alignment_length_p[k];
	//printf("processing bam qc batch p id: %i, strand (+): %i, map quality: %i, alignment length: %i\n", bam_qc_batch_p->id, bam_qc_batch_p->strand_counter_p[k], bam_qc_batch_p->map_quality_p[k], bam_qc_batch_p->alignment_length_p[k]);
      }

      for (int k=0; k<bam_qc_batch_p->num_alignments; k++) {
	bam_qc_report.map_error_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[MISMATCHES] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[MISMATCHES] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
	bam_qc_report.map_deletion_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[D] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[D] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
	bam_qc_report.map_insertion_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[I] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[I] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
	bam_qc_report.map_matching_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[EQUAL] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[EQUAL] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;	
      }
      
      sprintf(log_message, "Thread-RESULTS: ....processing for batch %i done !\n",bam_qc_batch_p->id);
      LOG_DEBUG(log_message);
      
      // free ALL memory
      //
      bam_qc_batch_free(bam_qc_batch_p, false);

      if (time_flag) { stop_timer(t1_result, t2_result, result_time); }
    } //end of if-else
  
    // getting gpus and cpus thread status
    //
    pthread_mutex_lock(&gpus_thread_alive_lock);
    gpus_alive = gpus_thread_alive;
    pthread_mutex_unlock(&gpus_thread_alive_lock);
    
    pthread_mutex_lock(&cpus_thread_alive_lock);
    cpus_alive = cpus_thread_alive;
    pthread_mutex_unlock(&cpus_thread_alive_lock);
  
    // next batch...
    //
    //bam_qc_batch_p = bam_qc_batch_list_remove(&bam_qc_batch_list);
    item_p = list_remove_item(&bam_qc_batch_list);
    
    if (item_p != NULL) {
      bam_qc_batch_p = (bam_qc_batch_t*) item_p->data_p;
    } else {
      //bam_qc_batch_p = NULL;
      break;
    }
  } // end of batch loop

  printf("bam_qc_report.num_alignments: %i, strand (+): %i, strand (-): %i\n", bam_qc_report.num_alignments, bam_qc_report.strand_counter, (bam_qc_report.num_alignments - bam_qc_report.strand_counter));

  if (time_flag) { start_timer(t1_result); }

  // calculate mean quality and mean length per alignment
  //
  printf("bam_qc_report.mean_read_quality: %i, num_alignments: %i, mean_quality: %i\n", bam_qc_report.mean_map_quality, bam_qc_report.num_alignments, (bam_qc_report.mean_map_quality / bam_qc_report.num_alignments));
  printf("bam_qc_report.mean_alignment_length: %i, num_alignments: %i, mean_alignment_length: %i\n", bam_qc_report.mean_alignment_length, bam_qc_report.num_alignments, (bam_qc_report.mean_alignment_length / bam_qc_report.num_alignments));
     
  bam_qc_report.mean_map_quality /= bam_qc_report.num_alignments;
  bam_qc_report.mean_alignment_length /= bam_qc_report.num_alignments;
  
  if (time_flag) { stop_timer(t1_result, t2_result, result_time); }		

  // and finally, print qc report, data files and graphs
  // when cpu data is ready (cpus_alive = 0)

  while (cpus_alive>0) {
    //printf("waiting....\n");
    sched_yield();

    // Delay for a bit
    //struct timespec ts;
    //ts.tv_sec = 0;
    //ts.tv_nsec = 100000000;
    //nanosleep (&ts, NULL);	
    
    usleep(10000);
    
    pthread_mutex_lock(&cpus_thread_alive_lock);
    cpus_alive = cpus_thread_alive;
    pthread_mutex_unlock(&cpus_thread_alive_lock);
  }  

  if (time_flag) { start_timer(t1_result); }

  bam_qc_report.num_mappings_histogram = qc_mapping_counter_p->num_mappings_histogram;
  bam_qc_report.mean_paired_end_distance = qc_mapping_counter_p->mean_paired_end_distance;
  
  if (time_flag) { stop_timer(t1_result, t2_result, result_time); }

  // --------------- D E B U G ----------------
  
//   printf("--------------- D E B U G ----------------\n");
//   for (int i=0; i<=(MAX_MAPPING_COUNT_IN_HISTOGRAM + 1); i++) {
//     printf("bam_qc_report.num_mappings_histogram[%i]: %i\n", i, bam_qc_report.num_mappings_histogram[i]);
//   }
//   printf("bam_qc_report.mean_paired_end_distance: %ld\n\n", bam_qc_report.mean_paired_end_distance);
  
  // --------------- D E B U G ----------------

  if (time_flag) { start_timer(t1_reporting); }
  generate_report(bam_qc_report, input_p->filename, input_p->base_quality, input_p->report_directory, 1);
  if (time_flag) { stop_timer(t1_reporting, t2_reporting, reporting_time); }

  if (time_flag) { stop_timer(t1_results_server, t2_results_server, results_server_time); }
  LOG_DEBUG("Thread-RESULTS: END\n");

  // exiting....
  //
  pthread_exit(0);
   
}


/*
      QC BAM FILE
*/

void qc_bam_file(size_t batch_size, int batch_list_size, int gpu_num_threads, int gpu_num_blocks, int cpu_num_threads, int base_quality, int max_distance_size, char* input_filename, char* output_directory, char* gff_filename) {

/*  bam_data_core_t* d_core_data_aux_p;
  CUDA_SAFE_CALL( hipHostAlloc((void**) &d_core_data_aux_p, (unsigned int) (num_alignments + 1) * sizeof(bam_data_core_t), 0) );*/
  
  // number of GPUs is obtained, and initializes the number of GPU threads 'alive'
  //
  int num_gpu_devices;
  hipError_t cudaResultCode = hipGetDeviceCount(&num_gpu_devices);
  if (cudaResultCode != hipSuccess) {
    num_gpu_devices = 0;
  }
  gpus_thread_alive = num_gpu_devices;
  
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  if (!prop.canMapHostMemory) {
   LOG_FATAL("device does not support MapHostMemory\n");   
  }
  
  //initializing bam_data_batch_list, bam_qc_batch_list and qc_mapping_counter
  //
  //bam_data_batch_list_t bam_data_batch_list_gpu;
  //bam_data_batch_list_t bam_data_batch_list_cpu;
 
  //bam_data_batch_list_init(&bam_data_batch_list_gpu, 0);
  //bam_data_batch_list_init(&bam_data_batch_list_cpu, 0);
  //bam_qc_batch_list_init(&bam_qc_batch_list);
  
  list_t bam_data_batch_list_gpu;
  list_t bam_data_batch_list_cpu;

  if (num_gpu_devices > 0) {    
    list_init("bam_data_batch_list_cpu", num_gpu_devices, batch_list_size, &bam_data_batch_list_cpu);
  } else {
    list_init("bam_data_batch_list_cpu", 1, batch_list_size, &bam_data_batch_list_cpu);
  }
  list_init("bam_data_batch_list_gpu", 1, batch_list_size, &bam_data_batch_list_gpu);
  list_init("bam_qc_batch_list", 1, batch_list_size, &bam_qc_batch_list);

  qc_mapping_counter_t qc_mapping_counter;	//PASS TO THE CPU AND RESULTS THREADS AND BUILD THE MUTEX HANDLERS
  qc_mapping_counter_init(&qc_mapping_counter);

  // multi-threads
  //  
  bam_reader_t* bam_reader_p = bam_reader_by_batch_new(input_filename, batch_size, base_quality, &bam_data_batch_list_gpu, LIST_INSERT_MODE);
  bam_reader_start(bam_reader_p);
  //num_alignments = bam_reader_join(bam_reader_p);

  // some local variables
  //
  void* r;
  
  // multi-threads
  //
  pthread_t* qc_calc_server_thread_p = (pthread_t*) malloc(((num_gpu_devices == 0) ? 1 : num_gpu_devices) * sizeof(pthread_t));
  pthread_t* cpus_server_thread_p = (pthread_t*) malloc(cpu_num_threads * sizeof(pthread_t));
  pthread_t results_server_thread;

  // calling GPU threads to process the bam data,
  // but first, prepare input parameter
  //
  int i;
  qc_calc_server_input_t** qc_calc_server_input_p = (qc_calc_server_input_t**) calloc(num_gpu_devices, sizeof(qc_calc_server_input_t*));
  
  if (num_gpu_devices > 0) {
    for (i=0; i < num_gpu_devices; i++) {
      qc_calc_server_input_p[i] = (qc_calc_server_input_t*) calloc(1, sizeof(qc_calc_server_input_t));
    }
  } else {
    qc_calc_server_input_p[0] = (qc_calc_server_input_t*) calloc(1, sizeof(qc_calc_server_input_t));
  }

  if (num_gpu_devices > 0) {	// GPU implementacion
    for (i=0; i < num_gpu_devices; i++) {
	qc_calc_server_input_p[i]->num_gpu_devices = num_gpu_devices;
	qc_calc_server_input_p[i]->cpu_num_threads = 0;
	qc_calc_server_input_p[i]->gpu_device_id[0] = i;
	qc_calc_server_input_p[i]->gpu_num_blocks = gpu_num_blocks;
	qc_calc_server_input_p[i]->gpu_num_threads = gpu_num_threads;
	qc_calc_server_input_p[i]->gpu_batch_list_p = &bam_data_batch_list_gpu;
	qc_calc_server_input_p[i]->cpu_batch_list_p = &bam_data_batch_list_cpu;
	//bam_data_batch_list_incr_producers(&bam_data_batch_list_cpu);
	pthread_create(&qc_calc_server_thread_p[i], NULL, qc_calc_server, (void*) qc_calc_server_input_p[i]);
    }
  } else {			// CPU implementacion
    qc_calc_server_input_p[0]->num_gpu_devices = 0;
    qc_calc_server_input_p[0]->cpu_num_threads = cpu_num_threads;
    qc_calc_server_input_p[0]->gpu_device_id[0] = 0;
    qc_calc_server_input_p[0]->gpu_num_blocks = 0;
    qc_calc_server_input_p[0]->gpu_num_threads = 0;
    qc_calc_server_input_p[0]->gpu_batch_list_p = &bam_data_batch_list_gpu;
    qc_calc_server_input_p[0]->cpu_batch_list_p = &bam_data_batch_list_cpu;
    //bam_data_batch_list_incr_producers(&bam_data_batch_list_cpu);
    pthread_create(&qc_calc_server_thread_p[0], NULL, qc_calc_server, (void*) &qc_calc_server_input_p[0]);
  }

  /*for (int i=0; i < num_gpu_devices; i++) {
   pthread_join(qc_calc_server_thread_p[i], &r);
  }*/  
  
  // calling CPU threads to process the bam data,
  //
  //for (int i=0; i < cpu_num_threads; i++) {
  cpus_server_input_t* cpus_server_input_p = (cpus_server_input_t*) calloc(1, sizeof(cpus_server_input_t));
  
  for (i=0; i < 1; i++) {
    cpus_server_input_p->cpu_num_threads = cpu_num_threads;
    cpus_server_input_p->max_distance_size = max_distance_size;
    cpus_server_input_p->cpu_batch_list_p = &bam_data_batch_list_cpu;
    cpus_server_input_p->qc_mapping_counter = &qc_mapping_counter;
    cpus_server_input_p->gff_filename = gff_filename;
    cpus_server_input_p->output_directory = output_directory;
    cpus_server_input_p->input_filename = input_filename;
      
    pthread_create(&cpus_server_thread_p[i], NULL, cpus_server, (void*) cpus_server_input_p);
  }
  
/*  
  // wait for all terminating
  //  
  for (int i=0; i < num_gpu_devices; i++) {
   pthread_join(qc_calc_server_thread_p[i], &r);
  }  
  
  for (int i=0; i < 1; i++) {
   pthread_join(cpus_server_thread_p[i], &r);
  } */

  // calling thread to process results from GPU,
  //
  results_server_input_t results_server_input;
  results_server_input.gpu_num_blocks = gpu_num_blocks;
  results_server_input.gpu_num_threads = gpu_num_threads;
  results_server_input.base_quality = base_quality;
  results_server_input.qc_mapping_counter = &qc_mapping_counter;
  results_server_input.filename = input_filename;
  results_server_input.report_directory = output_directory;
  pthread_create(&results_server_thread, NULL, results_server, (void*) &results_server_input);

  num_alignments = bam_reader_join(bam_reader_p);
  
  for (int i=0; i < num_gpu_devices; i++) {
   pthread_join(qc_calc_server_thread_p[i], &r);
   //pthread_detach(qc_calc_server_thread_p[i]);
  }
  free(qc_calc_server_thread_p);
  
  for (int i=0; i < 1; i++) {
   pthread_join(cpus_server_thread_p[i], &r);
   //pthread_detach(cpus_server_thread_p[i]);
  }
  free(cpus_server_thread_p);
  
  pthread_join(results_server_thread, &r);

  // free thread stuff and parameters
  
  if (num_gpu_devices > 0) {
    for (i=0; i < num_gpu_devices; i++) {
      free(qc_calc_server_input_p[i]);
    }
  } else {
    free(qc_calc_server_input_p[0]);
  }
  
  free(qc_calc_server_input_p);  
  free(cpus_server_input_p);
  bam_reader_free(bam_reader_p);

}

#endif /* QC_CU_ */