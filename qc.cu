#include "hip/hip_runtime.h"

#ifndef QC_CU
#define QC_CU

extern "C" {
    #include "bam_coverage.h"
    #include "bam_data_batch.h"
    #include "bam_data_batch_list.h"
    #include "bam_qc_batch.h"
    #include "bam_qc_report.h"
    #include "bam_reader.h"
    #include "commons.h"
    #include "file_utils.h"
    #include "gff_data.h"
    #include "gff_reader.h"
    #include "log.h"
    #include "qc.h"
    #include "qc_kernel_omp.h"
    #include "sam.h"
    #include "system_utils.h"
}

#include "qc_kernel_cuda.h"

/* ******************************************************
 *    		Private thread functions 		*
 * *****************************************************/

void* qc_calc_server(void* params_p);
void* cpus_server(void* params_p);
void* results_server(void* params_p);

/* **********************************************
 *    		Global variables 		*
 * *********************************************/

list_t bam_qc_batch_list;

int bam_batch_reader_alive = 1;
int gpus_thread_alive = 1;
int cpus_thread_alive = 1;

pthread_mutex_t gpus_thread_alive_lock = PTHREAD_MUTEX_INITIALIZER;
pthread_mutex_t cpus_thread_alive_lock = PTHREAD_MUTEX_INITIALIZER;

/* **********************************************
 *    		Extern calls	 		*
 * *********************************************/

extern void call_kernel_basic_stats(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, int* d_strand_counter_p, int* d_alignment_length_p, int* d_map_quality_p, int num_alignments);
extern void call_kernel_map_errors(dim3 dimGrid, dim3 dimBlock, bam_data_core_t* d_core_data_p, uint32_t* d_cigar_data_p, qc_alignment_t* d_qc_alignment_p, int num_alignments);

/* **********************************************************************
 *    		Private thread functions implementations 		*
 * *********************************************************************/

/* ******************************************************
 *    		QC calc server thread	 		*
 * *****************************************************/

void* qc_calc_server(void* params_p) {
    LOG_DEBUG("Thread-GPU: START\n");

    if (time_flag) {
        start_timer(t1_qc_calc_server);
    }

    qc_calc_server_input_t* input_p = (qc_calc_server_input_t*) params_p;

    int cpu_num_threads = input_p->cpu_num_threads;
    list_item_t* bam_data_batch_list_item_p = NULL;
    list_t* gpu_batch_list_p = input_p->gpu_batch_list_p;
    list_t* cpu_batch_list_p = input_p->cpu_batch_list_p;
    bam_data_batch_t* bam_data_batch_p = NULL;
    bam_qc_batch_t* bam_qc_batch_p = NULL;

    // variables for store output results in both CPU and GPU
    qc_alignment_t* qc_alignment_p;
    int* strand_counter_p;
    int* map_quality_p;
    int* alignment_length_p;

    bam_data_core_t* d_core_data_p;
    uint32_t* d_cigar_data_p;
    qc_alignment_t* d_qc_alignment_p;
    int* d_strand_counter_p;
    int* d_map_quality_p;
    int* d_alignment_length_p;

    // selecting GPU device
    CUDA_SAFE_CALL(hipSetDevice(input_p->gpu_device_id[0]));

    int reads_alive;

    reads_alive = list_get_writers(gpu_batch_list_p);


    while ((bam_data_batch_list_item_p = list_remove_item(gpu_batch_list_p)) != NULL) {
        LOG_DEBUG("Thread-GPU: waiting for batch....\n");

        //if (time_flag) { 
        //    start_timer(t1_gpu); 
        //}

        if ((time_flag) && (gpus_standby_time == 0.0)) {
            stop_timer(t1_active_reader, t1_active_gpus, gpus_standby_time);
        }

        char log_message[50];
        sprintf(log_message, "Thread-GPU: processing for batch %i....\n", bam_data_batch_list_item_p->id);
        LOG_DEBUG(log_message);

        number_of_batchs++;

        // allocation memory for output results
        bam_data_batch_p = (bam_data_batch_t*) bam_data_batch_list_item_p->data_p;
        int num_alignments = bam_data_batch_p->num_alignments;
        int num_blocks;

//cpu_num_threads = 1;
        if (cpu_num_threads == 0) {  // GPU implementation
            num_blocks = (num_alignments / input_p->gpu_num_threads) + 1;

            dim3 dimBlock(input_p->gpu_num_threads, 1, 1);
            dim3 dimGrid(num_blocks, 1, 1);

            strand_counter_p = (int*) calloc(num_blocks, sizeof(int));
            map_quality_p = (int*) calloc(num_blocks, sizeof(int));
            alignment_length_p = (int*) calloc(num_blocks, sizeof(int));
            qc_alignment_p = (qc_alignment_t*) calloc(bam_data_batch_p->num_alignments, sizeof(qc_alignment_t));

            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_core_data_p, (unsigned int)(num_alignments + 1) * sizeof(bam_data_core_t), 0) );
            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_strand_counter_p, (unsigned int) num_blocks * sizeof(int), 0) );
            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_map_quality_p, (unsigned int) num_blocks * sizeof(int), 0) );
            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_alignment_length_p, (unsigned int) num_blocks * sizeof(int), 0) );
            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_cigar_data_p, (unsigned int) bam_data_batch_p->num_cigar_operations * sizeof(uint32_t), 0) );
            CUDA_SAFE_CALL( hipHostAlloc((void**) &d_qc_alignment_p, (unsigned int) num_alignments * sizeof(qc_alignment_t), 0) );

            CUDA_SAFE_CALL( hipMemcpy(d_core_data_p, bam_data_batch_p->core_data_p, (num_alignments + 1) * sizeof(bam_data_core_t), hipMemcpyHostToDevice) );
            CUDA_SAFE_CALL( hipMemcpy(d_cigar_data_p, bam_data_batch_p->cigar_data_p, bam_data_batch_p->num_cigar_operations * sizeof(uint32_t), hipMemcpyHostToDevice) );

            CUDA_START_TIMER();
            call_kernel_basic_stats(dimGrid, dimBlock, d_core_data_p, d_strand_counter_p, d_map_quality_p, d_alignment_length_p, num_alignments);
            call_kernel_map_errors(dimGrid, dimBlock, d_core_data_p, d_cigar_data_p, d_qc_alignment_p, num_alignments);
            CUDA_STOP_TIMER();

            // copy result from GPU (GPU -> CPU)
            CUDA_SAFE_CALL( hipMemcpy(strand_counter_p, d_strand_counter_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
            CUDA_SAFE_CALL( hipMemcpy(map_quality_p, d_map_quality_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
            CUDA_SAFE_CALL( hipMemcpy(alignment_length_p, d_alignment_length_p, num_blocks * sizeof(int), hipMemcpyDeviceToHost) );
            CUDA_SAFE_CALL( hipMemcpy(qc_alignment_p, d_qc_alignment_p, num_alignments * sizeof(qc_alignment_t), hipMemcpyDeviceToHost) );

            // free device memory
            CUDA_SAFE_CALL( hipHostFree(d_core_data_p) );
            CUDA_SAFE_CALL( hipHostFree(d_cigar_data_p) );
            CUDA_SAFE_CALL( hipHostFree(d_strand_counter_p) );
            CUDA_SAFE_CALL( hipHostFree(d_map_quality_p) );
            CUDA_SAFE_CALL( hipHostFree(d_qc_alignment_p) );
        } else {
            // accumulation of partial results is only made once
            num_blocks = 1;

            strand_counter_p = (int*) calloc(num_blocks, sizeof(int));
            map_quality_p = (int*) calloc(num_blocks, sizeof(int));
            alignment_length_p = (int*) calloc(num_blocks, sizeof(int));
            qc_alignment_p = (qc_alignment_t*) calloc(bam_data_batch_p->num_alignments, sizeof(qc_alignment_t));

            if (time_flag) {
                start_timer(t1_gpu);
            }
            cpu_bam_qc_basic_stats(bam_data_batch_p->core_data_p, strand_counter_p, map_quality_p, alignment_length_p, num_alignments, cpu_num_threads);
            cpu_bam_qc_map_errors(bam_data_batch_p->core_data_p, bam_data_batch_p->cigar_data_p, qc_alignment_p, num_alignments);
            if (time_flag) {
                stop_timer(t1_gpu, t2_gpu, gpu_time);
            }
        }

        //if (time_flag) { 
        //    stop_timer(t1_gpu, t2_gpu, gpu_time); 
        //}

        // create a new qc_batch object
        bam_qc_batch_p = (bam_qc_batch_t*) malloc(sizeof(bam_qc_batch_t));
        bam_qc_batch_p->id = bam_data_batch_list_item_p->id;
        bam_qc_batch_p->num_alignments = bam_data_batch_p->num_alignments;
        bam_qc_batch_p->num_blocks = num_blocks;
        bam_qc_batch_p->qc_alignment_p = qc_alignment_p;
        bam_qc_batch_p->strand_counter_p = strand_counter_p;
        bam_qc_batch_p->map_quality_p = map_quality_p;
        bam_qc_batch_p->alignment_length_p = alignment_length_p;
        //bam_qc_batch_p->alignments_p = bam_data_batch_list_item_p->alignments_p;

        // and insert it into the bam_qc_batch_list
        list_item_t* item_p = list_item_new(bam_qc_batch_p->id, 0, bam_qc_batch_p);
        list_insert_item(item_p, &bam_qc_batch_list);

        // copy the the current batch item to the cpu batch list in order to perform CPU qc operations
        list_insert_item(bam_data_batch_list_item_p, cpu_batch_list_p);

        sprintf(log_message, "Thread-GPU:...processing for batch %i done !\n", bam_data_batch_list_item_p->id);
        LOG_DEBUG(log_message);

        //if (time_flag) { stop_timer(t1_gpu, t2_gpu, gpu_time); }
    } // end of external while loop

    pthread_mutex_lock(&gpus_thread_alive_lock);
    gpus_thread_alive--;
    pthread_mutex_unlock(&gpus_thread_alive_lock);

    list_decr_writers(cpu_batch_list_p);
    list_decr_writers(&bam_qc_batch_list);

    if (time_flag) {
        stop_timer(t1_qc_calc_server, t2_qc_calc_server, qc_calc_server_time);
    }
    LOG_DEBUG("Thread-GPU: END\n");

    // exiting...
    pthread_exit(0);
}

/* ******************************************************
 *    		QC calc server thread	 		*
 * *****************************************************/

void* cpus_server(void* params_p) {
    double coverage_time = 0.0;
    struct timeval t1_coverage, t2_coverage;

    LOG_DEBUG("Thread-CPU: START\n");

    if (time_flag) {
        start_timer(t1_cpus_server);
    }

    //initialize str_coverage_matrix
    str_coverage_matrix_init();

    cpus_server_input_t* input_p = (cpus_server_input_t*) params_p;

    qc_mapping_counter_t* qc_mapping_counter_p = (qc_mapping_counter_t*) input_p->qc_mapping_counter;
    int max_distance_size = input_p->max_distance_size;
    int cpu_num_threads =  input_p->cpu_num_threads;
    bam_data_batch_t* bam_data_batch_p = NULL;
    list_item_t* bam_data_batch_list_item_p = NULL;
    list_t* cpu_batch_list_p = input_p->cpu_batch_list_p;
    char* gff_filename = input_p->gff_filename;
    char* output_directory = input_p->output_directory;
    char* input_filename = input_p->input_filename;

    // delete previous coverage file to append new data
    bam_coverage_counter_delete_file(output_directory, input_filename);

    // variables for store intermediate and output results in both CPU and GPU
    qc_hash_t* qc_hash_p = (qc_hash_t*) qc_hash_new(QC_HASH_LENGTH);

    // variables for coverage (regions data)
    bam_chromosome_coverage_t bam_chromosome_coverage[num_of_chromosomes];

    for (int j = 0; j < num_of_chromosomes; j++) {
        bam_chromosome_coverage_init(&bam_chromosome_coverage[j]);
    }

    gff_data_t* gff_data_p = gff_data_new(gff_filename);

    int gpus_alive;

    gpus_alive = list_get_writers(cpu_batch_list_p);

    while ((bam_data_batch_list_item_p = list_remove_item(cpu_batch_list_p)) != NULL) {
        if ((time_flag) && (cpus_standby_time == 0.0)) {
            stop_timer(t1_active_reader, t1_active_cpus, cpus_standby_time);
        }

        char log_message[50];
        sprintf(log_message, "Thread-CPU: processing for batch %i....\n", bam_data_batch_list_item_p->id);
        LOG_DEBUG(log_message);

        // allocation memory for output results
        bam_data_batch_p = (bam_data_batch_t*) bam_data_batch_list_item_p->data_p;
        int num_alignments = bam_data_batch_p->num_alignments;
        int cpu_num_threads = input_p->cpu_num_threads;

        if (time_flag) {
            start_timer(t1_cpu);
        }

        char* id_seq;
        int tid, start_coordinate, seq_length;
        short int paired_end;
        bam_data_core_t* core_data_p;

        for (int i = 0; i < bam_data_batch_p->num_alignments; i++) {
            id_seq = &(bam_data_batch_p->id_seq_data_p[bam_data_batch_p->core_data_p[i].id_seq_index]);

            core_data_p = &(bam_data_batch_p->core_data_p[i]);

            tid = core_data_p->chromosome;
            start_coordinate = core_data_p->start_coordinate;
            seq_length = core_data_p->alignment_length;
            paired_end = core_data_p->paired_end;

            qc_hash_insert_alignment(qc_hash_p, id_seq, tid, start_coordinate, seq_length, paired_end);
        }

        //if (time_flag) { 
        //    start_timer(t1_coverage); 
        //}

        if (gff_data_batch_in_region(bam_data_batch_p, gff_data_p) != 0) {
            bam_coverage_compute(bam_data_batch_p, bam_chromosome_coverage, gff_data_p, output_directory, input_filename, cpu_num_threads);
        }

        //if (time_flag) { 
        //    stop_timer(t1_coverage, t2_coverage, coverage_time); 
        //}

        if (time_flag) {
            stop_timer(t1_cpu, t2_cpu, cpu_time);
        }

        sprintf(log_message, "Thread-CPU:...processing for batch %i done !\n", bam_data_batch_list_item_p->id);
        LOG_DEBUG(log_message);
 
        // free the current batch item if all processing with the batch is performed
        bam_data_batch_free((bam_data_batch_t*) bam_data_batch_list_item_p->data_p);
        list_item_free(bam_data_batch_list_item_p);
 
        // ask again for reads server status
        gpus_alive = list_get_writers(cpu_batch_list_p);
    } // end of external while loop

    // print the last counters
    bam_coverage_counter_mark_to_print(bam_chromosome_coverage, true);
    bam_coverage_counter_print(bam_chromosome_coverage, output_directory, input_filename);

    //qc_hash_list_print(qc_hash_p->qc_hash_list_p);

    //calculate over the qc hash table to obtain:
    //    - Mean distance between paired ends
    //    - Histogram of mappings per reads

    unsigned long mean_paired_end_distance = 0;

    if (time_flag) {
        start_timer(t1_cpu);
    }

    qc_hash_perform_calculations(qc_hash_p, qc_mapping_counter_p, &mean_paired_end_distance, max_distance_size, cpu_num_threads); //--

    qc_mapping_counter_p->mean_paired_end_distance = mean_paired_end_distance;
    if (time_flag) {
        stop_timer(t1_cpu, t2_cpu, cpu_time);
    }

    //free qc hash structure, gff data and chromosome coverage
    for (int j = 0; j < num_of_chromosomes; j++) {
        bam_chromosome_coverage_clear(&bam_chromosome_coverage[j]);
    }

    qc_hash_free(qc_hash_p, true);
    gff_data_free(gff_data_p);

    // --------------- D E B U G ----------------

    printf("--------------- D E B U G ----------------\n");

    for (int i = 0; i <= (MAX_MAPPING_COUNT_IN_HISTOGRAM + 1); i++) {
        printf("qc_mapping_counter_p->num_mappings_histogram[%i]: %i\n", i, qc_mapping_counter_p->num_mappings_histogram[i]);
    }
    printf("mean_paired_end_distance: %ld\n\n", qc_mapping_counter_p->mean_paired_end_distance);

    printf("--------------- D E B U G ----------------\n");

    // --------------- D E B U G ----------------

    pthread_mutex_lock(&cpus_thread_alive_lock);
    cpus_thread_alive--;
    pthread_mutex_unlock(&cpus_thread_alive_lock);

    if (time_flag) {
        stop_timer(t1_cpus_server, t2_cpus_server, cpus_server_time);
    }

    LOG_DEBUG("Thread-CPU: END\n");

    // exiting...
    pthread_exit(0);
}

/* ******************************************************
 *    		Results server thread	 		*
 * *****************************************************/

void* results_server(void* params_p) {
    LOG_DEBUG("Thread-RESULTS: START\n");

    if (time_flag) {
        start_timer(t1_results_server);
    }

    results_server_input_t* input_p = (results_server_input_t*) params_p;

    // variables for storing qc report information
    bam_qc_report_t bam_qc_report;
    memset(&bam_qc_report, 0, sizeof(bam_qc_report_t));

    qc_mapping_counter_t* qc_mapping_counter_p = (qc_mapping_counter_t*) input_p->qc_mapping_counter;
    int nb_total_threads = input_p->gpu_num_blocks * input_p->gpu_num_threads;
    int base_quality = input_p->base_quality;
    int i, alignments;

    // go through the results_batch list, and process it
    // take and remove the first item, and so on...
    list_item_t* item_p = NULL;
    bam_qc_batch_t* bam_qc_batch_p = NULL;
    int gpus_alive, cpus_alive;

    // getting gpus thread status
    pthread_mutex_lock(&gpus_thread_alive_lock);
    gpus_alive = gpus_thread_alive;
    pthread_mutex_unlock(&gpus_thread_alive_lock);

    pthread_mutex_lock(&cpus_thread_alive_lock);
    cpus_alive = cpus_thread_alive;
    pthread_mutex_unlock(&cpus_thread_alive_lock);

    //iteration until not NULL is returned, then process batch
    while ((item_p = list_remove_item(&bam_qc_batch_list)) != NULL) {
        bam_qc_batch_p = (bam_qc_batch_t*) item_p->data_p;
        //printf("while... gpus_alive: %i, cpus_alive: %i, bam_qc_batch_p is NULL: %i\n", gpus_alive, cpus_alive, (bam_qc_batch_p == NULL) ? 1:0);
        if ((time_flag) && (results_standby_time == 0.0)) {
            stop_timer(t1_active_reader, t1_active_results, results_standby_time);
        }
        if (time_flag) {
            start_timer(t1_result);
        }

        char log_message[50];
        sprintf(log_message, "Thread-RESULTS: processing for bam batch %i....\n", bam_qc_batch_p->id);
        LOG_DEBUG(log_message);

        // result processing batch per batch
        alignments = bam_qc_batch_p->num_alignments;
        bam_qc_report.num_alignments += alignments;

        for (int k = 0; k < bam_qc_batch_p->num_blocks; k++) {
            bam_qc_report.strand_counter += bam_qc_batch_p->strand_counter_p[k];
            bam_qc_report.mean_map_quality += bam_qc_batch_p->map_quality_p[k];
            bam_qc_report.mean_alignment_length += bam_qc_batch_p->alignment_length_p[k];
        }

        for (int k = 0; k < bam_qc_batch_p->num_alignments; k++) {
            bam_qc_report.map_error_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[MISMATCHES] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[MISMATCHES] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
            bam_qc_report.map_deletion_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[D] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[D] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
            bam_qc_report.map_insertion_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[I] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[I] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
            bam_qc_report.map_matching_histogram[(bam_qc_batch_p->qc_alignment_p[k].counters[EQUAL] <= MAX_MAP_ERRORS_IN_HISTOGRAM) ? bam_qc_batch_p->qc_alignment_p[k].counters[EQUAL] : (MAX_MAP_ERRORS_IN_HISTOGRAM + 1)]++;
        }

        sprintf(log_message, "Thread-RESULTS: ....processing for batch %i done !\n", bam_qc_batch_p->id);
        LOG_DEBUG(log_message);

        // free ALL memory
        bam_qc_batch_free(bam_qc_batch_p, false);

        if (time_flag) {
            stop_timer(t1_result, t2_result, result_time);
        }


        // getting gpus and cpus thread status
        pthread_mutex_lock(&gpus_thread_alive_lock);
        gpus_alive = gpus_thread_alive;
        pthread_mutex_unlock(&gpus_thread_alive_lock);

        pthread_mutex_lock(&cpus_thread_alive_lock);
        cpus_alive = cpus_thread_alive;
        pthread_mutex_unlock(&cpus_thread_alive_lock);
    } // end of batch loop

    printf("bam_qc_report.num_alignments: %lu, strand (+): %i, strand (-): %i\n", bam_qc_report.num_alignments, bam_qc_report.strand_counter, (bam_qc_report.num_alignments - bam_qc_report.strand_counter));

    if (time_flag) {
        start_timer(t1_result);
    }

    //calculate mean quality and mean length per alignment
    if (bam_qc_report.num_alignments > 0) {
        printf("bam_qc_report.mean_read_quality: %lu, num_alignments: %lu, mean_quality: %i\n", bam_qc_report.mean_map_quality, bam_qc_report.num_alignments, (bam_qc_report.mean_map_quality / bam_qc_report.num_alignments));
        printf("bam_qc_report.mean_alignment_length: %lu, num_alignments: %lu, mean_alignment_length: %i\n", bam_qc_report.mean_alignment_length, bam_qc_report.num_alignments, (bam_qc_report.mean_alignment_length / bam_qc_report.num_alignments));
        bam_qc_report.mean_map_quality /= bam_qc_report.num_alignments;
        bam_qc_report.mean_alignment_length /= bam_qc_report.num_alignments;
    } else {
        printf("bam_qc_report.mean_read_quality: %lu, num_alignments: %lu, mean_quality: 0\n", bam_qc_report.mean_map_quality, bam_qc_report.num_alignments);
        printf("bam_qc_report.mean_alignment_length: %lu, num_alignments: %lu, mean_alignment_length: 0\n", bam_qc_report.mean_alignment_length, bam_qc_report.num_alignments);
        bam_qc_report.mean_map_quality = 0;
        bam_qc_report.mean_alignment_length = 0;
    }

    if (time_flag) {
        stop_timer(t1_result, t2_result, result_time);
    }

    // and finally, print qc report, data files and graphs
    // when cpu data is ready (cpus_alive = 0)
    while (cpus_alive > 0) {
        sched_yield();

        usleep(10000);

        pthread_mutex_lock(&cpus_thread_alive_lock);
        cpus_alive = cpus_thread_alive;
        pthread_mutex_unlock(&cpus_thread_alive_lock);
    }

    if (time_flag) {
        start_timer(t1_result);
    }

    bam_qc_report.num_mappings_histogram = qc_mapping_counter_p->num_mappings_histogram;
    bam_qc_report.mean_paired_end_distance = qc_mapping_counter_p->mean_paired_end_distance;

    if (time_flag) {
        stop_timer(t1_result, t2_result, result_time);
    }

    if (time_flag) {
        start_timer(t1_reporting);
    }
    generate_report(bam_qc_report, input_p->filename, input_p->base_quality, input_p->report_directory, 1);
    if (time_flag) {
        stop_timer(t1_reporting, t2_reporting, reporting_time);
    }

    if (time_flag) {
        stop_timer(t1_results_server, t2_results_server, results_server_time);
    }
    LOG_DEBUG("Thread-RESULTS: END\n");

    // exiting...
    pthread_exit(0);
}

/* **************************************************************
 *    		Public functions implementations 		*
 * *************************************************************/

void qc_bam_file(size_t batch_size, int batch_list_size, int gpu_num_threads, int gpu_num_blocks, int cpu_num_threads, int base_quality, int max_distance_size, char* input_filename, char* output_directory, char* gff_filename) {
    // number of GPUs is obtained, and initializes the number of GPU threads 'alive'
    int num_gpu_devices;
    hipError_t cudaResultCode = hipGetDeviceCount(&num_gpu_devices);
    if (cudaResultCode != hipSuccess) {
        num_gpu_devices = 0;
    }
    gpus_thread_alive = num_gpu_devices;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (!prop.canMapHostMemory) {
        LOG_FATAL("device does not support MapHostMemory\n");
    }

    //initializing bam_data_batch_list_gpu, bam_data_batch_list_cpu and bam_qc_batch_list
    list_t bam_data_batch_list_gpu;
    list_t bam_data_batch_list_cpu;

    if (num_gpu_devices > 0) {
        list_init("bam_data_batch_list_cpu", num_gpu_devices, batch_list_size, &bam_data_batch_list_cpu);
    } else {
        list_init("bam_data_batch_list_cpu", 1, batch_list_size, &bam_data_batch_list_cpu);
    }
    list_init("bam_data_batch_list_gpu", 1, batch_list_size, &bam_data_batch_list_gpu);
    list_init("bam_qc_batch_list", ((num_gpu_devices > 0) ? num_gpu_devices : 1), batch_list_size, &bam_qc_batch_list);

    //initializing qc_mapping_counter
    qc_mapping_counter_t qc_mapping_counter;
    qc_mapping_counter_init(&qc_mapping_counter);

    //multi-threads
    bam_reader_t* bam_reader_p = bam_reader_by_batch_new(input_filename, batch_size, base_quality, &bam_data_batch_list_gpu, LIST_INSERT_MODE);
    bam_reader_start(bam_reader_p);

    //some local variables
    void* r;

    // multi-threads
    pthread_t* qc_calc_server_thread_p = (pthread_t*) malloc(((num_gpu_devices == 0) ? 1 : num_gpu_devices) * sizeof(pthread_t));
    pthread_t* cpus_server_thread_p = (pthread_t*) malloc(cpu_num_threads * sizeof(pthread_t));
    pthread_t results_server_thread;

    //calling GPU threads to process the bam data,
    //but first, prepare input parameter
    int i;
    qc_calc_server_input_t** qc_calc_server_input_p = (qc_calc_server_input_t**) calloc(num_gpu_devices, sizeof(qc_calc_server_input_t*));

    if (num_gpu_devices > 0) {
        for (i = 0; i < num_gpu_devices; i++) {
            qc_calc_server_input_p[i] = (qc_calc_server_input_t*) calloc(1, sizeof(qc_calc_server_input_t));
        }
    } else {
        qc_calc_server_input_p[0] = (qc_calc_server_input_t*) calloc(1, sizeof(qc_calc_server_input_t));
    }

    if (num_gpu_devices > 0) { //GPU implementacion
        for (i = 0; i < num_gpu_devices; i++) {
            qc_calc_server_input_p[i]->num_gpu_devices = num_gpu_devices;
            qc_calc_server_input_p[i]->cpu_num_threads = 0;
            qc_calc_server_input_p[i]->gpu_device_id[0] = i;
            qc_calc_server_input_p[i]->gpu_num_blocks = gpu_num_blocks;
            qc_calc_server_input_p[i]->gpu_num_threads = gpu_num_threads;
            qc_calc_server_input_p[i]->gpu_batch_list_p = &bam_data_batch_list_gpu;
            qc_calc_server_input_p[i]->cpu_batch_list_p = &bam_data_batch_list_cpu;
            pthread_create(&qc_calc_server_thread_p[i], NULL, qc_calc_server, (void*) qc_calc_server_input_p[i]);
        }
    } else {   //CPU implementacion
        qc_calc_server_input_p[0]->num_gpu_devices = 0;
        qc_calc_server_input_p[0]->cpu_num_threads = cpu_num_threads;
        qc_calc_server_input_p[0]->gpu_device_id[0] = 0;
        qc_calc_server_input_p[0]->gpu_num_blocks = 0;
        qc_calc_server_input_p[0]->gpu_num_threads = 0;
        qc_calc_server_input_p[0]->gpu_batch_list_p = &bam_data_batch_list_gpu;
        qc_calc_server_input_p[0]->cpu_batch_list_p = &bam_data_batch_list_cpu;
        pthread_create(&qc_calc_server_thread_p[0], NULL, qc_calc_server, (void*) &qc_calc_server_input_p[0]);
    }

    // calling CPU threads to process the bam data,
    cpus_server_input_t* cpus_server_input_p = (cpus_server_input_t*) calloc(1, sizeof(cpus_server_input_t));

    for (i = 0; i < 1; i++) {
        cpus_server_input_p->cpu_num_threads = cpu_num_threads;
        cpus_server_input_p->max_distance_size = max_distance_size;
        cpus_server_input_p->cpu_batch_list_p = &bam_data_batch_list_cpu;
        cpus_server_input_p->qc_mapping_counter = &qc_mapping_counter;
        cpus_server_input_p->gff_filename = gff_filename;
        cpus_server_input_p->output_directory = output_directory;
        cpus_server_input_p->input_filename = input_filename;

        pthread_create(&cpus_server_thread_p[i], NULL, cpus_server, (void*) cpus_server_input_p);
    }

    //calling thread to process results from GPU,
    results_server_input_t results_server_input;
    results_server_input.gpu_num_blocks = gpu_num_blocks;
    results_server_input.gpu_num_threads = gpu_num_threads;
    results_server_input.base_quality = base_quality;
    results_server_input.qc_mapping_counter = &qc_mapping_counter;
    results_server_input.filename = input_filename;
    results_server_input.report_directory = output_directory;
    pthread_create(&results_server_thread, NULL, results_server, (void*) &results_server_input);
    num_alignments = bam_reader_join(bam_reader_p);

    for (int i = 0; i < num_gpu_devices; i++) {
        pthread_join(qc_calc_server_thread_p[i], &r);
    }
    free(qc_calc_server_thread_p);

    for (int i = 0; i < 1; i++) {
        pthread_join(cpus_server_thread_p[i], &r);
    }

    free(cpus_server_thread_p);
    pthread_join(results_server_thread, &r);

    //free thread stuff and parameters
    if (num_gpu_devices > 0) {
        for (i = 0; i < num_gpu_devices; i++) {
            free(qc_calc_server_input_p[i]);
        }
    } else {
        free(qc_calc_server_input_p[0]);
    }

    free(qc_calc_server_input_p);
    free(cpus_server_input_p);
    bam_reader_free(bam_reader_p);
}

#endif /* QC_CU */
